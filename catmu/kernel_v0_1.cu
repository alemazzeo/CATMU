#include "hip/hip_runtime.h"
// """
// Este kernel combina la utilizacion de la memoria compartida para almacenar
// la lista de posiciones con la interpolación de la PSF LUT que aplica la 
// Unidad de Mapeo de Texturas (TMU) para calcular la convolución.
//
// La PSF LUT puede indicar un factor de escala respecto de la unidad utilizada
// para la imagen resultante. Por ejemplo, si cada pixel de la PSF mide A y
// los pixels de la imagen resultante miden B, deberá indicarse:
//
// psf.w_size = 
// """ 
#include "convolution_lut.h"

__global__ void lutKernel2D(sImage2d image, Positions2d pos, sPSF psf, 
                            hipTextureObject_t texPSF){

    // Worker ID
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    int idk = blockDim.x * threadIdx.y + threadIdx.x;

    extern __shared__ float shared[];

    if (blockDim.x * blockDim.y >= pos.n * 2){
        if (idk < pos.n * 2){
            shared[idk] = pos.data[idk];
        }
    }
    else{
        int m = pos.n * 2 / blockDim.x * blockDim.y;
        for (int i = 0; i < m; i++){
            if (idk * m + i < pos.n * 2){
                shared[idk * m + i] = pos.data[idk * m + i];
            }
        }
    }

    __syncthreads();

    float px, py, pixel;
    float factor_x, factor_y;

    factor_x = image.pixel_width / psf.pixel_width / psf.width;
    factor_y = image.pixel_height / psf.pixel_height / psf.height;

    // Condition for valid work
    if (idx < image.width && idy < image.height) {
        pixel = 0;
        for (int i = 0; i < pos.n; i+=2){
            px = (float) (idx-shared[i*2]) * factor_x + 0.5;
            py = (float) (idy-shared[i*2+1]) * factor_y + 0.5;
            pixel += tex2D<float>(texPSF, px, py);
        }
        image.data[idy * image.width + idx] = pixel;
    }
}

int launch_kernels(sImage2d * h_image, Positions2d * h_pos, sPSF * h_psf, 
                   sImage2d * d_image, Positions2d * d_pos, 
                   hipTextureObject_t * texObj){
    
    info_print("Kernel file: %s\n", __FILE__);

    info_print("Grid and block sizes:\n");
    dim3 dimBlock(16, 16);
    dim3 dimGrid((h_image->width  + dimBlock.x - 1) / dimBlock.x,
                 (h_image->height + dimBlock.y - 1) / dimBlock.y);

    info_print("dimGrid: %dx%d\n", dimGrid.x, dimGrid.y);
    info_print("dimBlock: %dx%d\n", dimBlock.x, dimBlock.y);

    // Tamaño para la memoria compartida
    size_t sm = h_pos->n * sizeof(float) * 2;

    info_print("Executing kernels\n");
    lutKernel2D <<<dimGrid, dimBlock, sm>>> (*d_image, *d_pos, *h_psf, *texObj);
    info_print("Kernels finished\n");
    CUDA_CHECK_ERROR(return err);
    return 0;
}