#include "convolution_lut.h"

int set_texture2d(hipArray * cuArray, sPSF * psf, 
                   hipTextureObject_t * texObj, bool normalized){

    info_print("Allocate CUDA array in device memory\n");
    hipChannelFormatDesc channelDesc =
               hipCreateChannelDesc(32, 0, 0, 0,
                                     hipChannelFormatKindFloat);
    hipMallocArray(&cuArray, &channelDesc, psf->width, psf->height);

    CUDA_CHECK_ERROR(return err);

    info_print("Copy to device memory sPSF data from host\n");
    /*hipMemcpyToArray(cuArray, 0, 0, psf->data, 
                      (psf->width * psf->height) * sizeof(float), 
                      hipMemcpyHostToDevice);*/

    hipMemcpy2DToArray(cuArray, 0, 0, psf->data,
                        psf->width * sizeof(float),
                        psf->width * sizeof(float),
                        psf->height,
                        hipMemcpyHostToDevice);

    CUDA_CHECK_ERROR(return err);

    info_print("Specify texture\n");
    struct hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(resDesc));
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = cuArray;
    CUDA_CHECK_ERROR(return err);

    info_print("Specify texture object parameters\n");
    struct hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(texDesc));
    texDesc.addressMode[0]   = hipAddressModeBorder;
    texDesc.addressMode[1]   = hipAddressModeBorder;
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.readMode         = hipReadModeElementType;
    texDesc.normalizedCoords = 0;
    CUDA_CHECK_ERROR(return err);

    info_print("Create texture object\n");
    hipCreateTextureObject(texObj, &resDesc, &texDesc, NULL);
    CUDA_CHECK_ERROR(return err);

    return 0;

}

int free_texture(hipArray * cuArray, hipTextureObject_t * texObj){
    info_print("Destroy texture object\n");
    hipDestroyTextureObject(*texObj);
    info_print("Free texture memory\n");
    hipFreeArray(cuArray);
    return 0;
}