#include "hip/hip_runtime.h"
// """
// Este kernel de prueba escribe sobre la imagen objetivo
// la identificación del thread utilizado
// """ 
#include "convolution_lut.h"

__global__ void lutKernel2D(sImage2d image, Positions2d pos, sPSF psf, 
                            hipTextureObject_t texPSF){

    // Worker ID
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    size_t idy = blockIdx.y * blockDim.y + threadIdx.y;

    // Condition for valid work
    if (idx < image.width && idy < image.height) {
        image.data[idy * image.width + idx] = idy * image.width + idx;
    }
}

int launch_kernels(sImage2d * h_image, Positions2d * h_pos, sPSF * h_psf, 
                   sImage2d * d_image, Positions2d * d_pos, 
                   hipTextureObject_t * texObj){
    
    info_print("Kernel file: %s\n", __FILE__);

    info_print("Grid and block sizes:\n");
    dim3 dimBlock(16, 16);
    dim3 dimGrid((h_image->width  + dimBlock.x - 1) / dimBlock.x,
                 (h_image->height + dimBlock.y - 1) / dimBlock.y);

    info_print("dimGrid: %dx%d\n", dimGrid.x, dimGrid.y);
    info_print("dimBlock: %dx%d\n", dimBlock.x, dimBlock.y);

    info_print("Executing kernels\n");
    lutKernel2D <<<dimGrid, dimBlock>>> (*d_image, *d_pos, *h_psf, *texObj);
    info_print("Kernels finished\n");
    return 0;
}